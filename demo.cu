#include "hip/hip_runtime.h"
#include <windows.h>

#define width 1920
#define height 1080

typedef unsigned int uint;

//////////////////////////////////////////////////////////////////////

__device__ float dot(float3 a, float3 b)
{
	return (a.x * b.x + a.y * b.y + a.z * b.z);
}

__device__ float length(float3 v)
{
	return sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
}

__device__ float3 normalize(float3 v)
{
	float n = 1.0f / sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
	return  make_float3(n * v.x, n * v.y, n * v.z);
}

__device__ float3 reflect( float3 i, float3 n )
{
	float d = (n.x * i.x + n.y * i.y + n.z * i.z);
	return make_float3(i.x - 2.0f * n.x * d, i.y - 2.0f * n.y * d, i.z - 2.0f * n.z * d);
}

__device__ float clamp(float x, float a, float b)
{
	return fmaxf(a, fminf(b, x));
}

__device__ float4 texelFetch(float4 *sampler, uint x, uint y, int dx, int dy)  //fragCoord (x,y) and offset(dx,dy)
{
	uint q = (height - (y + dy) - 1) * width + (x + dx);
	return sampler[q];
}

__device__ float3 texelFetch3(float4 *sampler, uint x, uint y, int dx, int dy)
{
	uint q = (height - (y + dy) - 1) * width + (x + dx);
	float4 p = sampler[q];
	return make_float3(p.x, p.y, p.z);
}

//////////////////////////////////////////////////////////////////////

__global__ void BufferA(float4 *fragColor, float4 *buffer, float mx, float my)
{
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;
	uint id = (height - y - 1) * width + x;
	float kspring = 2.0f;
	float dspring = 1.0f;
	float mass = 20.0f;
	float dt = 1.0f;
	float2 mouse = make_float2(mx, my);
	float4 currentState = texelFetch(buffer, x, y, 0, 0);
	float force = 0.0f;
	float2 m = make_float2((x - mouse.x)*(x - mouse.x), (y - mouse.y)*(y - mouse.y));
	if (m.x < 5.f && m.y < 5.f) force -= 200.0f;           
	if (x<2 || x>width-2 || y<2 || y>height-2) return;
	for (int i=-1; i<=1; i+=2)
	{
		for (int j=-1; j<=1; j+=2)
		{
			float4 neighborState = texelFetch(buffer, x, y, i, j);
			if (x<4 || x>width-4 || y<4 || y>height-4) neighborState = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
			float deltaP = neighborState.x - currentState.x;
			float deltaV = neighborState.y - currentState.y;
			force += kspring * deltaP + dspring * deltaV;
		}
	}
	float acceleration = force / mass;
	float velocity = acceleration * dt + currentState.y;
	float position = velocity * dt + currentState.x;
	fragColor[id] = make_float4(position, velocity, acceleration, 1.0f);
}

__global__ void BufferB(float4 *fragColor, float4 *buffer)
{
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;
	uint id = (height - y - 1) * width + x;
	float4 state = texelFetch(buffer, x, y, 0, 0);
	float position = saturate(state.x / 1024.0f + 0.5f);
	fragColor[id] = make_float4(0.0f, 0.3f, (255.0f*position)/255.0f + 0.5f, 1.0f);
}

__global__ void Image(uchar4 *fragColor, float4 *buffer)
{
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;
	uint id = (height - y - 1) * width + x;
	if (x<2 || x>width-2 || y<2 || y>height-2) return;
	float a = length(texelFetch3(buffer, x, y,  1, 0));  
	float b = length(texelFetch3(buffer, x, y, -1, 0));  
	float c = length(texelFetch3(buffer, x, y,  0, 1));  
	float d = length(texelFetch3(buffer, x, y,  0,-1));  
	float3 normal = normalize(make_float3((a-b)*height, (c-d)*height, 5.0f));
	float3 light = normalize(make_float3(1.f,1.f,2.f));
	float diffuse = clamp(dot(normal,light),0.3f,1.0f);
	float specular = powf(clamp(dot(reflect(light,normal),make_float3(0.f,0.f,-1.f)),0.0f,1.0f), 32.0f);
	float4 color = make_float4(buffer[id].x*diffuse+specular, buffer[id].y*diffuse+specular, buffer[id].z*diffuse+specular, 1.0f);
	fragColor[id] = make_uchar4(saturate(color.z)*255, saturate(color.y)*255, saturate(color.x)*255, 255);
}

//////////////////////////////////////////////////////////////////////

static LRESULT CALLBACK WindowProc(HWND hWnd, UINT uMsg, WPARAM wParam, LPARAM lParam)
{
	if (uMsg==WM_CLOSE || uMsg==WM_DESTROY || (uMsg==WM_KEYDOWN && wParam==VK_ESCAPE))
	{
		PostQuitMessage(0); return 0;
	}
	else
	{
		return DefWindowProc(hWnd, uMsg, wParam, lParam);
	}
}

int WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, LPSTR lpCmdLine, int nCmdShow)
{
	ShowCursor(0);
	int exit = 0;
	MSG msg;
	WNDCLASS win = {CS_OWNDC|CS_HREDRAW|CS_VREDRAW, WindowProc, 0, 0, 0, 0, 0, (HBRUSH)(COLOR_WINDOW+1), 0, "CUDA Demo"};
	RegisterClass(&win);
	HDC hdc = GetDC(CreateWindowEx(0, win.lpszClassName, "CUDA Demo", WS_VISIBLE|WS_POPUP, 0, 0, width, height, 0, 0, 0, 0));
	float4 *bufferA, *bufferB;
	uchar4 *image; 
	hipMalloc( (void**)&bufferA, width*height*sizeof(float4) );
	hipMalloc( (void**)&bufferB, width*height*sizeof(float4) );
	hipMalloc( (void**)&image, width*height*sizeof(uchar4) );
	dim3 block(8, 8);
	dim3 grid(width/8, height/8);
	BITMAPINFO bmi = {{sizeof(BITMAPINFOHEADER),width,height,1,32,BI_RGB,0,0,0,0,0},{0,0,0,0}};
	static unsigned char host[width*height*4];
	POINT point;
	DWORD S = GetTickCount();
	while (!exit)
	{
		while(PeekMessage(&msg, 0, 0, 0, PM_REMOVE))
		{
			if( msg.message==WM_QUIT ) exit = 1;
			TranslateMessage( &msg );
			DispatchMessage( &msg );
		}
		if (((GetTickCount() - S) % 3 == 0) && (GetAsyncKeyState(VK_LBUTTON)& 0x8000)) GetCursorPos(&point);
		BufferA<<<grid, block>>>(bufferA, bufferA, point.x, point.y);
		BufferB<<<grid, block>>>(bufferB, bufferA);
		Image<<<grid, block>>>(image, bufferB);
		hipMemcpy(host, image, width * height * sizeof(uchar4), hipMemcpyDeviceToHost);
		StretchDIBits(hdc,0,0,width,height,0,0,width,height,host,&bmi,DIB_RGB_COLORS,SRCCOPY);
	}
	hipFree(bufferA);
	hipFree(bufferB);
	hipFree(image);
	return 0;
}